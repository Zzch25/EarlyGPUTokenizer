/*
 *This puppy is gonna go up on github since I'll be using
 *it in my next project so before I forget...
 *
 *(c) Zachary Job
 *Sharing is caring, use requires that I am contacted and
 *give permission. OR the code can be displayed without
 *modification including original comments for educational
 *purposes!
 *
 *fBThreadMgr.cu
 *4/15/2015
 */

#include <stdio.h>
#include <stdlib.h>

#include "../Definitions/cpuCall.h"
#include "../Definitions/gpuCall.h"

#include "../settings.h"

#include "localDef.h"

/*
 * Launches a thread to run the parser and launch kernels
 * in parallel
 * 
 * @PARAM: The arguments
 */
void *fBThreadMgr(void *arg)
{
	int		
		reading, laststate, TID, pages,
		
		*feedbufferA, *feedbufferB,
		*histogram,
	
		wrTOP = 1, wrLOW = 1, last = 0,
		feedBytes;
	long
		readIdx = 0;
		
	hipStream_t stream;
	
	// Get back pointers and values - Long for said pointers
	TID = (int)((long *)arg)[0];
	histogram = (int *)(long)((long *)arg)[1];
	pages = (int)((long *)arg)[2] - TID;
	feedBytes = sizeof(int) * FEED_BY;
	TID += 1;
	
	//enable 
	hipSetDevice(0);
	//set for parrallel work
	hipStreamCreate(&stream);
	
	//Allocate the feed buffers
	//feedbuffer = (int *)malloc(feedBytes);
	hipHostMalloc((void**)&feedbufferA, feedBytes);
	hipHostMalloc((void**)&feedbufferB, feedBytes);

	// if GPU memory allocation failed, report an error message
	if(!feedbufferA || !feedbufferB)
		fprintf(stderr, "CudaMalloc thread reports failure\n"), exit(-7);
	
	// Setup the execution configuration
	dim3 threads(THREADS_PER_BUFF, 1, 1);
	dim3 blocks(FEED_BUFFERS, 1, 1);
	
	//Move to the correct thread offset in the file
	if(pages > 0)
		laststate = seekThreadOffs(&readIdx, TID);
	reading = (int)(laststate > 0);
 
	// CPU batching with co-processing
	while(reading)
	{
		reading = fillBuffer(feedbufferA, TID, &laststate, &pages, 
								&wrTOP, &wrLOW, &last, &readIdx);
		hipStreamSynchronize(stream);
		wattoken_kernel<<<blocks,threads,0,stream>>>(feedbufferA, histogram);
		
		if(reading)
		{
			reading = fillBuffer(feedbufferB, TID, &laststate, &pages, 
									&wrTOP, &wrLOW, &last, &readIdx);
			hipStreamSynchronize(stream);
			wattoken_kernel<<<blocks,threads,0,stream>>>(feedbufferB, histogram);
		}
	}
 	
	//End work, clean
	hipStreamDestroy(stream);
	hipHostFree(feedbufferB);
	hipHostFree(feedbufferA);	
	free(arg);

	return NULL;
}