#include "hip/hip_runtime.h"
/*
 *This puppy is gonna go up on github since I'll be using
 *it in my next project so before I forget...
 *
 *(c) Zachary Job
 *All rights reserved, I am not liable for damages.
 *Re-distributable and use with my permission contact me
 *at zachjob@verizon.net
 *
 *Presentable without modification including comments 
 *for educational purposes
 *
 *wattoken_kernel.cu
 *4/15/2015
 *
 *This is designed for a minimum architecture of the
 *GK104 specifications (specifically the k520). 
 *Otherwise go away.
 *
 *Tokenize the input buffer and update the global histogram
 */

#include <stdio.h> 

#include "../Definitions/defs.h"

#include "../../../settings.h"

/*
 *
 * The most intense unrolling you may ever see, behold...
 *
 * The goal is to utilize EVERY drop of memory that is lbuffer
 * >global to regs
 * >process regs and move to shared
 * >waterfall process
 *		>parallel reduction where their content will only survive if they are the first
 *			occurrence of a key from left to right in the buffer
 * EXAMPLE... REMEMBER, high memory volume is required for effectiveness unlike here
 *	buffer of 8x4 w/ 16 threads
							
						ENTIRE SH MEM IS PROCESSED
						******** 24 threads check 24 elements each and modify their count
						********
						********
						********
						
						EX
						*123****->	*123****
						********->	********
						********->	********
						******15->	******05
						
 * >write shared to global with all threads, also using the SH 0 bucket to recklessly
 *	remove contention because life in the fast lane has caused me to loose my mind 
 *
 */
__global__  void wattoken_kernel(int *bufferLoc, int *cpuHist)
{
	__shared__	int shmem[FEED_BY];	//The waterfall buffer where computed hashes are stored
	
	char							//iterators and offset chunk into shared
			trip;				    //the buffer flush indicator and the waterfall trip
	int		
			idxL, j, i,
			shchnk, count;							//hold the result of a hash
			//shoffs;
	
	/*
	idxL = threadIdx.x * FDBY_LD;
	
	//Fetch everything to memory in every other position
	for(i = idxL; i < idxL + FDBY_LD; i++)
		shmem[i] = bufferLoc[i];
	
	__syncthreads();
	*/
	
	//unrolled - see above for comments
	
	idxL = threadIdx.x * FDBY_LD;
	
	shmem[idxL] = bufferLoc[idxL];
	shmem[idxL + 1] = bufferLoc[idxL + 1];
	shmem[idxL + 2] = bufferLoc[idxL + 2];
	shmem[idxL + 3] = bufferLoc[idxL + 3];
	shmem[idxL + 4] = bufferLoc[idxL + 4];
	shmem[idxL + 5] = bufferLoc[idxL + 5];
	shmem[idxL + 6] = bufferLoc[idxL + 6];
	shmem[idxL + 7] = bufferLoc[idxL + 7];
	shmem[idxL + 8] = bufferLoc[idxL + 8];
	shmem[idxL + 9] = bufferLoc[idxL + 9];
	shmem[idxL + 10] = bufferLoc[idxL + 10];
	shmem[idxL + 11] = bufferLoc[idxL + 11];
	shmem[idxL + 12] = bufferLoc[idxL + 12];
	shmem[idxL + 13] = bufferLoc[idxL + 13];
	shmem[idxL + 14] = bufferLoc[idxL + 14]; 
	shmem[idxL + 15] = bufferLoc[idxL + 15];
	shmem[idxL + 16] = bufferLoc[idxL + 16];
	shmem[idxL + 17] = bufferLoc[idxL + 17];
	shmem[idxL + 18] = bufferLoc[idxL + 18];
	shmem[idxL + 19] = bufferLoc[idxL + 19];
	shmem[idxL + 20] = bufferLoc[idxL + 20];
	shmem[idxL + 21] = bufferLoc[idxL + 21];
	shmem[idxL + 22] = bufferLoc[idxL + 22];
	shmem[idxL + 23] = bufferLoc[idxL + 23];
	shmem[idxL + 24] = bufferLoc[idxL + 24]; 
	shmem[idxL + 25] = bufferLoc[idxL + 25]; 
	shmem[idxL + 26] = bufferLoc[idxL + 26]; 
	shmem[idxL + 27] = bufferLoc[idxL + 27]; 
	shmem[idxL + 28] = bufferLoc[idxL + 28]; 
	shmem[idxL + 29] = bufferLoc[idxL + 29]; 
	shmem[idxL + 30] = bufferLoc[idxL + 30]; 
	shmem[idxL + 31] = bufferLoc[idxL + 31]; 
	shmem[idxL + 32] = bufferLoc[idxL + 32]; 
	shmem[idxL + 33] = bufferLoc[idxL + 33]; 
	shmem[idxL + 34] = bufferLoc[idxL + 34]; 
	shmem[idxL + 35] = bufferLoc[idxL + 35]; 
	shmem[idxL + 36] = bufferLoc[idxL + 36]; 
	shmem[idxL + 37] = bufferLoc[idxL + 37]; 
	shmem[idxL + 38] = bufferLoc[idxL + 38]; 
	shmem[idxL + 39] = bufferLoc[idxL + 39]; 
	shmem[idxL + 40] = bufferLoc[idxL + 40]; 
	shmem[idxL + 41] = bufferLoc[idxL + 41]; 
	shmem[idxL + 42] = bufferLoc[idxL + 42]; 
	shmem[idxL + 43] = bufferLoc[idxL + 43]; 
	shmem[idxL + 44] = bufferLoc[idxL + 44]; 
	shmem[idxL + 45] = bufferLoc[idxL + 45]; 
	shmem[idxL + 46] = bufferLoc[idxL + 46]; 
	shmem[idxL + 47] = bufferLoc[idxL + 47];
	
	__syncthreads();
	
	
	for(shchnk = 0, trip = 1; shchnk < FEED_BY; shchnk += THREADS_PER_BUFF, trip = 1)
	{
		//index
		idxL = threadIdx.x + shchnk;
		//get value of index
		j = shmem[idxL];
		//occurrences
		count = 0;
		
		//Warp target code. Each element in the shared buffer is checked
		//per thread with this calculation. Instances are shifted
		//backwards as the priority potential lessens as you go
		//down the buffer. Waterfall reduction is my best analogy
		//via zeroing greater indexes of already existing values
		//
		//  SEE FUNCTION COMMENT EXAMPLE 
		//

		for(i = shchnk, trip = 1; i < THREADS_PER_BUFF + shchnk; i++)
			count += (int)(shmem[i] == j), trip &= (char)((!((i < idxL) && shmem[i] == j)) || (i >= idxL));
		
		//Flush if bad result
		j *= trip;
		count *= trip;
		
		//Now the tricky part. Multiple blocks will be running. Four
		//is the intended amount for the k520. Testing must be performed
		//I may end up using only four feeders and have four cpu threads
		//process the results into the main hashmap. However, this is
		//a single test kernel. Once integrated these concerns will be
		//dealt with
		
		//The home histogram 0 position will be recklessly written. That
		//is the point. This is meant to allow concurrency.
		//
		//cannot use j, could have been flushed
		
		// Hash
		j += (j << 12);
		j ^= (j >> 22);
		j += (j << 4);
		j ^= (j >> 9);
		j += (j << 10);
		j ^= (j >> 2);
		j += (j << 7);
		j ^= (j >> 12);
		j &= LANG_BITS_MSK;
		
		if(trip == 1)
			atomicAdd(&cpuHist[j], count);
	}
	
	//unrolled - see above for comments
	/*
	for(shchnk = 0, trip = 1; shchnk < FEED_BY; shchnk += THREADS_PER_BUFF, trip = 1)
	{
		idxL = threadIdx.x + shchnk;
		j = shmem[idxL];
		count = 0;
		
		//0
		count += (int)(shmem[shchnk] == j), trip &= (char)((!((shchnk < idxL) && shmem[shchnk] == j)) || shchnk >= idxL);
		shoffs = 1 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 2 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 3 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 4 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 5 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 6 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 7 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 8 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 9 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 10 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 11 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 12 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 13 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 14 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 15 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 16 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 17 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 18 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 19 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 20 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 21 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 22 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 23 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 24 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 25 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 26 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 27 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 28 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 29 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 30 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 31 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 32 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 33 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 34 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 35 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 36 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 37 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 38 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 39 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 40 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 41 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 42 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 43 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 44 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 45 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 46 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 47 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 48 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 49 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 50 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 51 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 52 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 53 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 54 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 55 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 56 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 57 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 58 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 59 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 60 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 61 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 62 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 63 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 64 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 65 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 66 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 67 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 68 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 69 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 70 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 71 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 72 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 73 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 74 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 75 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 76 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 77 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 78 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 79 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 80 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 81 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 82 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 83 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 84 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 85 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 86 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 87 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 88 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 89 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 90 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 91 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 92 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 93 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 94 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 95 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 96 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 97 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 98 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 99 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 100 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 101 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 102 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 103 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 104 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 105 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 106 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 107 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 108 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 109 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 110 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 111 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 112 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 113 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 114 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 115 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 116 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 117 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 118 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 119 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 120 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 121 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 122 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 123 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 124 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 125 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 126 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 127 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 128 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 129 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 130 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 131 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 132 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 133 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 134 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 135 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 136 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 137 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 138 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 139 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 140 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 141 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 142 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 143 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 144 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 145 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 146 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 147 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 148 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 149 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 150 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 151 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 152 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 153 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 154 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 155 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 156 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 157 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 158 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 159 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 160 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 161 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 162 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 163 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 164 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 165 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 166 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 167 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 168 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 169 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 170 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 171 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 172 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 173 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 174 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 175 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 176 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 177 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 178 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 179 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 180 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 181 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 182 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 183 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 184 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 185 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 186 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 187 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 188 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 189 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 190 + shchnk;
		count += (int)(shmem[shoffs] == j), trip &= (char)((!((shoffs < idxL) && shmem[shoffs] == j)) || shoffs >= idxL);
		shoffs = 191 + shchnk;
		count += (int)(shmem[shoffs] == j);
		
		// Hash
		j += (j << 12);
		j ^= (j >> 22);
		j += (j << 4);
		j ^= (j >> 9);
		j += (j << 10);
		j ^= (j >> 2);
		j += (j << 7);
		j ^= (j >> 12);
		j &= LANG_BITS_MSK;
		
		j *= trip;
		count *= trip;
		
		if(trip == 1)
			atomicAdd(&cpuHist[j], count);
	}
	*/
	//end unroll - see above for comments
}