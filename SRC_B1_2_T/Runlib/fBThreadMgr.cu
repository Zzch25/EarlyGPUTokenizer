/*
 *This puppy is gonna go up on github since I'll be using
 *it in my next project so before I forget...
 *
 *(c) Zachary Job
 *Sharing is caring, use requires that I am contacted and
 *give permission. OR the code can be displayed without
 *modification including original comments for educational
 *purposes!
 *
 *fBThreadMgr.cu
 *4/15/2015
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include "../Definitions/cpuCall.h"
#include "../Definitions/gpuCall.h"

#include "../settings.h"

#include "localDef.h"

/*
 * Launches a thread to run the parser and launch kernels
 * in parallel
 * 
 * @PARAM: The arguments
 */
void *fBThreadMgr(void *arg)
{
	int		
		reading, laststate, TID, pages,
		
		*feedbufferA, *feedbufferB,
		*histogram,
	
		wrTOP = 1, wrLOW = 1, last = 0;
	long
		readIdx = 0;
	float 
		ms = 0;
		
	FILE
		*gpuTime, *cpuTime;
		
	struct timeval
		startCPU, endCPU;
		
	hipStream_t 
		stream;
		
	hipEvent_t 
		startGPU, stopGPU;

	// Get back pointers and values - Long for said pointers
	TID = (int)((long *)arg)[0];
	histogram = (int *)(long)((long *)arg)[1];
	pages = (int)((long *)arg)[2] - TID;
	feedbufferA = (int *)(long)((long *)arg)[3];
	feedbufferB = (int *)(long)((long *)arg)[4];
	TID += 1;
	
	//enable 
	hipSetDevice(0);
	//set for parrallel work
	hipStreamCreate(&stream);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	
	//Allocate the feed buffers
	//feedbuffer = (int *)malloc(feedBytes);
	//hipHostMalloc((void**)&feedbufferA, feedBytes);
	//hipHostMalloc((void**)&feedbufferB, feedBytes);

	// if GPU memory allocation failed, report an error message
	//if(!feedbufferA || !feedbufferB)
	//	fprintf(stderr, "CudaMalloc thread reports failure\n"), exit(-7);
	
	// Setup the execution configuration
	dim3 threads(THREADS_PER_BUFF, 1, 1);
	dim3 blocks(FEED_BUFFERS, 1, 1);
	
	if((cpuTime = fopen(CPU_TIME,"w")) == NULL)
		fprintf(stderr, "Dependency failure, %s could not open\n", CPU_TIME), exit(24);
	if((gpuTime = fopen(GPU_TIME,"w")) == NULL)
		fprintf(stderr, "Dependency failure, %s could not open\n", GPU_TIME), exit(24);
	
	//Move to the correct thread offset in the file
	if(pages > 0)
		laststate = seekThreadOffs(&readIdx, TID);
	reading = (int)(laststate > 0);
 
	// CPU batching with co-processing
	while(reading)
	{
		gettimeofday(&startCPU, NULL);
		reading = fillBuffer(feedbufferA, TID, &laststate, &pages, 
								&wrTOP, &wrLOW, &last, &readIdx);
		gettimeofday(&endCPU, NULL);
		fprintf(cpuTime, "%d\n",
		(int)((endCPU.tv_sec * 1000000 + endCPU.tv_usec) -
		(startCPU.tv_sec * 1000000 + startCPU.tv_usec)));
		
		hipStreamSynchronize(stream);
		hipEventRecord(startGPU);
		wattoken_kernel<<<blocks,threads,0,stream>>>(feedbufferA, histogram);
		hipEventRecord(stopGPU);
		hipEventSynchronize(stopGPU);
		hipEventElapsedTime(&ms, startGPU, stopGPU);
		fprintf(gpuTime, "%f\n", ms);
		ms = 0;
		if(reading)
		{
			gettimeofday(&startCPU, NULL);
			reading = fillBuffer(feedbufferB, TID, &laststate, &pages, 
									&wrTOP, &wrLOW, &last, &readIdx);
			gettimeofday(&endCPU, NULL);
			fprintf(cpuTime, "%d\n",
			(int)((endCPU.tv_sec * 1000000 + endCPU.tv_usec) -
			(startCPU.tv_sec * 1000000 + startCPU.tv_usec)));
			
			hipStreamSynchronize(stream);
			hipEventRecord(startGPU);
			wattoken_kernel<<<blocks,threads,0,stream>>>(feedbufferB, histogram);
			hipEventRecord(stopGPU);
			hipEventSynchronize(stopGPU);
			hipEventElapsedTime(&ms, startGPU, stopGPU);
			fprintf(gpuTime, "%f\n", ms);
			ms = 0;
		}
	}
 	
	//End work, clean
	hipStreamDestroy(stream);
	//hipHostFree(feedbufferB);
	//hipHostFree(feedbufferA);
	fclose(cpuTime);
	fclose(gpuTime);
	free(arg);

	return NULL;
}