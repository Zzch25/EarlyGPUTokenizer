/*
 *This puppy is gonna go up on github since I'll be using
 *it in my next project so before I forget...
 *
 *(c) Zachary Job
 *All rights reserved, I am not liable for damages.
 *Re-distributable and use with my permission contact me
 *at zachjob@verizon.net
 *
 *Presentable without modification including comments 
 *for educational purposes
 *
 *waterfall_tokenize.cu
 *4/15/2015
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <sys/time.h>

#include "Definitions/defs.h"
#include "Definitions/cpuCall.h"
#include "Definitions/gpuCall.h"

#include "settings.h"

/*
 * Executes a tokenize system using the exclusion parser
 *
 * Designed to coalesce and avoid all control statements.
 * This allows for lots of math tweaks to have a 0 divergence
 * kernel.
 *
 *
 * U S I N G   A   N A I V E   F I L E   B U F F E R... will be fixed
 */
int main(int argc, char **argv)
{
	
	//V A R I A B L E S//////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
	
	char
		*data, *cfg,
		
		// THIS IS A SUPER NAIVE FIX, CHUNKS ARE NEEDED
		*fileBuffer;
	
	int
		histBytes, feedBytes,
		optCmp, pages,
		
		*histogram, *feedbuffers[(HST_THRD << 1)],
		
		hostThreads,
				
		*memMapS, *dirtyBits, *memOffset,
		*memMapE, *trimArr,
		
		i;
	
	long
		*arg;
		
	struct tags_t
		*configTags;
		
	pthread_t
		threads[HST_THRD];
		
	struct timeval
		start, end;
		
	FILE
		*output;
	
	//C O N F I G U R A T I O N   A N D   I N S T A T A N T I A T I O N//////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
  
	if(argc < 5)
	{
		fprintf(stderr, "usage: ./program_name\nREQUIRED ARGS\n"
			   "FILE 1: Search configuration [search.cfg]\n"
			   "FILE 2: The tagged data to process [someData.type]\n"
			   "3: Task compute configuration, pre-compute,\n"
			   "or run with key pre-computation\n"
			   "filter [0:2]\n"
			   "EX) im_the_settings.cfg im_the_big_data.type 0\n"
			   "4: Container tags to be processed in a document EG 7000\n");
		return -1;
	}
	
	// Get the arguments
	optCmp 	= atoi(argv[3]);
	pages 	= atoi(argv[4]);
	data 	= strdup(argv[2]);
	cfg 	= strdup(argv[1]);
	
	if(optCmp < 0 || optCmp > 2)
	{
		fprintf(stderr,"Invalid compute option, run ./program_name for usage\n");
		return 2;
	}
	if(pages < 1)
	{
		fprintf(stderr,"Insufficient page count, run ./program_name for usage\n");
		return 3;
	}
	
	// General definitions
	hostThreads = HST_THRD;
	histBytes = sizeof(int) * HIST_SZ;
	feedBytes = sizeof(int) * FEED_BY;
	
	// Page locked histogram for access by CPU and GPU (unified memory)
	hipHostMalloc((void**)&histogram, histBytes);
	
	// if GPU memory allocation failed, report an error message
	if(!histogram)
		fprintf(stderr, "CudaMalloc host reports failure\n"), exit(-7);
	memset(histogram, 0, histBytes);
	
	for(i = 0; i < (HST_THRD << 1); i+=2)
	{
		hipHostMalloc((void**)&feedbuffers[i], feedBytes);
		hipHostMalloc((void**)&feedbuffers[i + 1], feedBytes);
		
		if(!feedbuffers[i] || !feedbuffers[i + 1])
			fprintf(stderr, "CudaMalloc host reports failure\n"), exit(-7);
	}
	
	// init the parser variables
	cfgInit(cfg, &memMapS, &memMapE, &dirtyBits, 
		&memOffset, optCmp, &configTags, &trimArr);
	
	// Enter if not precompute
	if(optCmp != 1)
	{
	
	//H O S T   T H R E A D I N G////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
	
		// Open the file to be parsed
		initThreadData(memMapS, memMapE, dirtyBits, memOffset,
						trimArr, configTags, &fileBuffer, data);
	
		fprintf(stderr, "Start GPU Compute\n");
		
		gettimeofday(&start, NULL);
	
		// Launch threads to co-process and batch information to the GPU
		while(hostThreads > 0)
		{
			arg = (long *)malloc(sizeof(long *) * 5);
			if(!arg)
				fprintf(stderr, "Malloc reports failure\n"), exit(-7);
			
			hostThreads--;
			arg[0] = (long)(HST_THRD - hostThreads);
			arg[1] = (long)histogram;
			arg[2] = (long)pages;
			arg[3] = (long)feedbuffers[hostThreads << 1];
			arg[4] = (long)feedbuffers[(hostThreads << 1) + 1];
			
			if(pthread_create(&threads[hostThreads],NULL,fBThreadMgr,(void *)arg) != 0)
				fprintf(stderr, "Host thread could not start\n"), exit(1);
		}
		
		// Reset
		hostThreads = HST_THRD;
		
		// Post join the threads for a clean death, a soldiers death
		while(hostThreads  == 8)
		{
			if(pthread_join(threads[--hostThreads],NULL) != 0)
				fprintf(stderr, "Host thread could not join\n"), exit(1);
		}
		
		gettimeofday(&end, NULL);
		
		// Everyone likes microsecondss
		fprintf(stderr, "TIME(us)::%ld\nEnd GPU Compute\n\n", 
												(end.tv_sec * 1000000 + end.tv_usec) -
												(start.tv_sec * 1000000 + start.tv_usec));
		
		// Open the file to write the histogram results to
		if((output = fopen(OUTPUT_HIST,"w")) == NULL)
			fprintf(stderr, "Dependency failure, %s could not open\n", OUTPUT_HIST), exit(24);
		// Write results
		for(i = 0; i < HIST_SZ; i++)
			if(histogram[i] != 0) fprintf(output, "%d,%d\n", i, histogram[i]);
		
	//C L E A N U P//////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////
		
		free(memMapS);
		free(memMapE);
		free(memOffset);
		free(dirtyBits);
		free(configTags);
		free(trimArr);
		free(data);
		free(cfg);
		free(fileBuffer);
		
		// free feedbuffers for all the streams
		for(i = 0; i < (HST_THRD << 1); i+=2)
		{
			hipHostFree(feedbuffers[i]);
			hipHostFree(feedbuffers[i + 1]);
		}
	}
	hipHostFree(histogram);
	
	return 0;
}